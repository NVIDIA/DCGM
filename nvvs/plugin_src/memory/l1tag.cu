#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "L1TagCuda.h"
#include "newrandom.h"

__device__ void ReportError
(
    const L1TagParams& params,
    const L1TagError& error
)
{
    unsigned64* pErrorCount = GetPtr<unsigned64*>(params.errorCountPtr);
    L1TagError* pErrorLog = GetPtr<L1TagError*>(params.errorLogPtr);

    // Increment the error counter
    unsigned64 errorIdx = atomicAdd(pErrorCount, 1);

    // Dump the failure if there is room in the error buffer
    if (errorIdx < params.errorLogLen)
    {
        pErrorLog[errorIdx] = error;
    }
}

// Given a 16bit offset and 16bit pseudorandom number, encode a 32 bit value
// from which we can easily extract the offset. This is done by storing the random value
// in the upper bits, then XOR-ing this value with the offset for the lower bits.
//
// This is superior to only storing the offset since the random data increases the likelihood
// of catching noise-dependent failures.
__device__ __forceinline__ uint32_t EncodeOffset(uint16_t offset, uint16_t rnd)
{
    return static_cast<uint32_t>(rnd << 16) | static_cast<uint32_t>(rnd ^ offset);
}

// In order to extract the offset from an encoded value, simply XOR the lower 16 bits with
// the upper 16 bits.
__device__ __forceinline__ uint16_t DecodeOffset(uint32_t value)
{
    return static_cast<uint16_t>(value >> 16) ^ static_cast<uint16_t>(value);
}

extern "C" __global__ void InitL1Data(const L1TagParams params)
{
    // Get resident SM ID 
    uint32_t smid;
    asm volatile ("mov.u32 %0, %%smid;" : "=r"(smid));

    // Each SM has its own data region
    const uint32_t smidDataBytes = params.sizeBytes / gridDim.x;
    uint32_t* buf = GetPtr<uint32_t*>(params.data + smid * smidDataBytes);

    // Init RNG (each SM data region will have the same data)
    unsigned64 s[2];
    InitRand<2>(s, params.randSeed + threadIdx.x);

    for (uint32_t i = threadIdx.x; i < smidDataBytes / sizeof(*buf); i += blockDim.x)
    {
        const uint16_t rnd = static_cast<uint16_t>(FastRand(s) >> 48);
        buf[i] = EncodeOffset(i, rnd);
    }
}

extern "C" __global__ void L1TagTest(const L1TagParams params)
{
    // Get SMID and thread info
    uint32_t smid;
    uint32_t warpid;
    uint32_t laneid;
    asm volatile ("mov.u32 %0, %%smid;"   : "=r"(smid));
    asm volatile ("mov.u32 %0, %%warpid;" : "=r"(warpid));
    asm volatile ("mov.u32 %0, %%laneid;" : "=r"(laneid));
    const uint32_t hwtid = laneid + warpid * warpSize;

    // Each SM has its own data region
    const uint32_t smidDataBytes = params.sizeBytes / gridDim.x;
    uint32_t* buf = GetPtr<uint32_t*>(params.data + smid * smidDataBytes);

    // Init RNG (each SM will use the same seed, for equivalent data accesses)
    unsigned64 s[2];
    InitRand<2>(s, params.randSeed + hwtid);
    uint32_t rnd = static_cast<uint32_t>(FastRand(s));

    // Run the test for the specified iterations
    for (uint64_t iter = 0; iter < params.iterations; iter++)
    {
        // We run the inner loop once for each offset into a cache line
        constexpr uint32_t lineNumElem = L1_LINE_SIZE_BYTES / sizeof(*buf);
        for (uint32_t lineOff = 0; lineOff < lineNumElem; lineOff++)
        {
            const uint16_t preLoadOff = lineOff + (hwtid * lineNumElem);
            const uint16_t randOff = rnd % (smidDataBytes / sizeof(*buf));
            uint32_t preLoadVal = 0;
            uint32_t randVal    = 0;

            // Fill up the L1 Cache
            __syncthreads();
            asm volatile("ld.global.ca.u32 %0, [%1];":"=r"(preLoadVal):"l"(buf + preLoadOff));
#if (SM_VER == 82)
            const bool doSecondRead = (hwtid + blockDim.x) < (smidDataBytes / L1_LINE_SIZE_BYTES);
            const uint16_t altPreLoadOff = preLoadOff + (blockDim.x * lineNumElem);
            uint32_t altPreLoadVal = 0;
            if (doSecondRead)
            {
                asm volatile("ld.global.ca.u32 %0, [%1];":"=r"(altPreLoadVal):"l"(buf + altPreLoadOff));
            }
#endif
            __syncthreads();

            // With the L1 cache fully loaded, randomly read data (RandomLoad)
            asm volatile("ld.global.ca.u32 %0, [%1];":"=r"(randVal):"l"(buf + randOff));

            // Check the values after all reads are complete. Since latency matters in this test
            // we don't want to waste any cycles that could instead be used on random L1 data loads.
            //
            // Of course, the compiler will still reorder non-memory instructions,
            // but this is better than nothing.
            __syncthreads();
            const uint16_t decodedPreLoad = DecodeOffset(preLoadVal);
            if (decodedPreLoad != preLoadOff)
            {
                const L1TagError err =
                {
                    TestStage::PreLoad, decodedPreLoad, preLoadOff,
                    iter, lineOff, smid, warpid, laneid
                };
                ReportError(params, err);
            }
#if (SM_VER == 82)
            if (doSecondRead)
            {
                const uint16_t altDecodedPreLoad = DecodeOffset(altPreLoadVal);
                if (altDecodedPreLoad != altPreLoadOff)
                {
                    const L1TagError err =
                    {
                        TestStage::PreLoad, altDecodedPreLoad, altPreLoadOff,
                        iter, lineOff, smid, warpid, laneid
                    };
                    ReportError(params, err);
                }
            }
#endif
            const uint16_t decodedRand = DecodeOffset(randVal);
            if (decodedRand != randOff)
            {
                const L1TagError err =
                {
                    TestStage::RandomLoad, decodedRand, randOff,
                    iter, lineOff, smid, warpid, laneid

                };
                ReportError(params, err);
            }

            // Always use a new random offset
            rnd = static_cast<uint32_t>(FastRand(s));
        }
    }
}
